#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#include "my_crs_matrix.h"

#include <hip/hip_runtime.h>
#include <hipsparse.h>

__global__ void cgkernel()
{
	
}

__host__ my_crs_2_cusparse(my_crs_matrix* A, hipsparseHandle_t cusparseHandle);
{
	int nnz = A->nz;
	int n = A->n;

	// Allocate memory for the CSR matrix
	int *ptr, *indices, *data;
	hipMalloc((void **)&ptr, (n+1) * sizeof(int));
	hipMalloc((void **)&indices, nnz * sizeof(int));
	hipMalloc((void **)&data, nnz * sizeof(PRECI_DT));

	// Copy data from host to device
	hipMemcpy(ptr, A->rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(indices, A->col, nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(data, A->val, nnz * sizeof(PRECI_DT), hipMemcpyHostToDevice);

	// Create the CSR matrix
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseHybMat_t csrMat;
	hipsparseCreateHybMat(&csrMat);
	hipsparseDcsr2hyb(cusparseHandle, n, n, descr, data, ptr, indices, csrMat, 0, HIPSPARSE_HYB_PARTITION_AUTO);
}

int main(void)
{
    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);
    
    my_crs_matrix* A = my_crs_read("test_subjects/");
	
    my_crs_2_cusparse(A, cusparseHandle);
    // Run, Time and copy data from CG
    clock_t t;
    t = clock();
    cgkernel<<<1,1>>>();

    hipDeviceSynchronize();
    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC;
    printf("cg took %f seconds\n", time_taken);
    hipMemcpy(h_x, d_x, sizeof(PRECI_DT)*size, hipMemcpyDeviceToHost);

    hipsparseDestroy(cusparseHandle);
    return 0;
  }
