#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include "../include/CuCG.h"

//#include "../include/my_crs_matrix.h"
#define PRECI_DT double 
#define PRECI_S "%lf "
#define PRECI_CUDA HIP_R_64F

typedef struct {
  hipsparseDnVecDescr_t desc;
  PRECI_DT*            val;
} my_cuda_vector;

typedef struct {
  hipsparseSpMatDescr_t desc;
  int n;
  int m;
  int nz;
  PRECI_DT *val;
  int *col;
  int *rowptr;
} my_cuda_csr_matrix;

__host__ void cusparse_conjugate_gradient(my_cuda_csr_matrix *A,
					  my_cuda_csr_matrix *M,
					  my_cuda_vector *b,
					  my_cuda_vector *x,
					  int max_iter,
					 PRECI_DT tolerance,
					  hipsparseHandle_t* handle,
					  hipblasHandle_t* handle_blas)

{
  int n = A->n;

  #ifdef ENABLE_TESTS
  PRECI_DT* onex = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* onez = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* oner = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* oneq = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* onep = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
	int* rowptr;
	int* col;
	PRECI_DT* val;
    val = (PRECI_DT*)malloc(sizeof(PRECI_DT)*A->nz);
    col = (int*)malloc(sizeof(int)*A->nz);
    rowptr = (int*)malloc(sizeof(int)*A->n+1);

    hipMemcpy(rowptr, M->rowptr, (A->n+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col, M->col, A->nz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(val, M->val, A->nz * sizeof(PRECI_DT), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
    printf("CUDA rowptr : ");
    for( int i = 0; i <= A->n; i++)
      printf("%d ",rowptr[i]);
    printf("\n");

    printf("CUDA col : ");
    for( int i = 0; i < A->nz; i++)
      printf("%d ",col[i]);
    printf("\n");

    printf("CUDA val : ");
    for( int i = 0; i < A->nz; i++)
      printf("%lf ",val[i]);
    printf("\n");
  #endif
  size_t pitch;


  // Make r vector
  my_cuda_vector *r_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *p_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *q_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *z_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));

  hipMallocPitch((void**)&r_vec->val,&pitch, n * sizeof(PRECI_DT), 1);
  hipMallocPitch((void**)&p_vec->val,&pitch, n * sizeof(PRECI_DT),1);
  hipMallocPitch((void**)&q_vec->val,&pitch, n * sizeof(PRECI_DT),1);
  hipMallocPitch((void**)&z_vec->val,&pitch, n * sizeof(PRECI_DT),1);

  PRECI_DT* h_rpqz = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  for(int i=0;i<n;i++) h_rpqz[i] = 1;

  hipMemcpy(r_vec->val, h_rpqz, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(p_vec->val, h_rpqz, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(q_vec->val, h_rpqz, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(z_vec->val, h_rpqz, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);

  hipsparseCreateDnVec(&r_vec->desc, n, r_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&p_vec->desc, n, p_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&q_vec->desc, n, q_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&z_vec->desc, n, z_vec->val,PRECI_CUDA);

  free(h_rpqz);

  hipblasStatus_t sb;
  
  PRECI_DT alpha = 1.0;
  PRECI_DT beta = 0.0;
  const double ne_one = -1.0;
  const double n_one = 1.0;
  const double one = 0.0;

  int iter = 0;

  PRECI_DT v = 0;
  PRECI_DT Rho = 0;
  PRECI_DT Rtmp = 0;

  PRECI_DT res_norm = 0;
  PRECI_DT init_norm = 0;
  PRECI_DT ratio = 0;

  
  double Tiny = 0.1e-28;
  double minus_alpha = 0.0;

  // x is already zero
  
  size_t bufferSizeMV;
  void* buff;
  hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, b->desc, &one, x->desc, PRECI_CUDA, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
  hipMalloc(&buff, bufferSizeMV);


  /*hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  printf("\INITIAL VEC CREATION\n x1 = %lf \t alpha= %lf \t beta= %lf "
	 "\n v "
	 "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
	 "ratio(%lf) > tolerance(%lf)\n\n\n",
	 iter, onex[0], alpha, beta, v, oner[0], onep[0], oneq[0], onez[0], ratio,
	 tolerance);*/


  //matvec(A,x,r);
  hipsparseSpMV(*handle,
	       HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
	       &n_one,//alpha
	       A->desc,//matrix
	       x->desc,//vector
	       &one,//beta
	       r_vec->desc,//answer
	       PRECI_CUDA,//data type
	       HIPSPARSE_MV_ALG_DEFAULT,//algorithm
	       buff//buffer
	       );
  //hipDeviceSynchronize();

  // r = b - r
  hipblasDaxpy(*handle_blas, n, &ne_one, r_vec->val, 1, b->val, 1);
  //hipDeviceSynchronize();
  hipblasDcopy(*handle_blas,n,b->val, 1, r_vec->val, 1);
  //hipDeviceSynchronize();

  // z = r
  if (M)
      //z = MT\(M\r);
      M=A;
  else
      // z = r
      hipblasDcopy(*handle_blas,n,r_vec->val, 1, z_vec->val, 1);
  //hipDeviceSynchronize();

  // p = z
  hipblasDcopy(*handle_blas,n,z_vec->val, 1, p_vec->val, 1);
  //hipDeviceSynchronize();
  hipblasDnrm2(*handle_blas, n, r_vec->val, 1, &res_norm);
  //hipDeviceSynchronize();
  init_norm = res_norm;
  ratio = 1.0;

  #ifdef ENABLE_TESTS
  hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);

  hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  /*printf("PREQUEL \n x1 = %lf \t alpha= %lf \t beta= %lf "
	 "\n v "
	 "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
	 "ratio(%lf) > tolerance(%e)\n\n\n",
	 iter, onex[0], alpha, beta, v, oner[0], onep[0], oneq[0], onez[0], ratio,
	 tolerance);*/
  #endif

  while (iter <= max_iter && ratio > tolerance)
    {
  #ifdef ENABLE_TESTS
      printf("\nITERATION %d\n",iter);
  #endif
      iter++;

      if (M)
          //z = MT\(M\r);
          M=A;
      else
          // z = r
          hipblasDcopy(*handle_blas,n,r_vec->val, 1, z_vec->val, 1);
      //hipDeviceSynchronize();
  #ifdef ENABLE_TESTS
      hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("z[1] = %lf\n",onez[1]);
  #endif

      // Rho = r z dot prod
      hipblasDdot(*handle_blas, n, r_vec->val, 1, z_vec->val, 1, &Rho);
      //hipDeviceSynchronize();
  #ifdef ENABLE_TESTS
      printf("Rho = %lf\n",Rho);
  #endif

      // p = z + (beta * p)
      // p = (beta * z) + p
      if (iter == 1)
	    {
	      hipblasDcopy(*handle_blas,n,z_vec->val, 1, p_vec->val, 1);
//	      hipDeviceSynchronize();
	    }
      else
	    {
	      beta = Rho / (v + Tiny);
	      hipblasDscal(*handle_blas, n, &beta, p_vec->val, 1);
	      hipblasDaxpy(*handle_blas, n, &n_one, z_vec->val, 1, p_vec->val, 1);
 	//      hipDeviceSynchronize();
	    }
  #ifdef ENABLE_TESTS
	    printf("beta = %lf\n",beta);
	    hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
	    printf("p[1] = %lf\n",onep[1]);
  #endif
	

      hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, p_vec->desc, &one, q_vec->desc, PRECI_CUDA, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
      hipMalloc(&buff, bufferSizeMV);

      hipsparseSpMV(*handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
        &n_one,//alpha
        A->desc,//matrix
        p_vec->desc,//vector
        &one,//beta
        q_vec->desc,//answer
        PRECI_CUDA,//data type
        HIPSPARSE_MV_ALG_DEFAULT,//algorithm
        buff//buffer
      );
//      hipDeviceSynchronize();

#ifdef ENABLE_TESTS
      hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("q[1] = %lf\n",oneq[1]);
#endif
      
      // Rtmp = p q dot prod
      hipblasDdot(*handle_blas, n, p_vec->val, 1, q_vec->val, 1, &Rtmp);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      printf("Rtmp = %lf\n",Rtmp);
#endif

      // v = r z dot prod
      hipblasDdot(*handle_blas, n, r_vec->val, 1, z_vec->val, 1, &v);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("v = %lf\n",v);
#endif

      //alpha
      alpha = Rho / (Rtmp + Tiny);
#ifdef ENABLE_TESTS
      printf("alpha = %lf\n",alpha);
#endif
      
      // x = x + alpha * p
      hipblasDaxpy(*handle_blas, n, &alpha, p_vec->val, 1, x->val, 1);
//      hipDeviceSynchronize();

#ifdef ENABLE_TESTS
      hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("x[1] = %lf\n",onex[1]);
#endif

      // r = r - alpha * q
      minus_alpha = -alpha;
      hipblasDaxpy(*handle_blas, n, &minus_alpha,q_vec->val,1,r_vec->val,1);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif

      Rho = 0.0;
      hipblasDnrm2(*handle_blas, n, r_vec->val, 1, &res_norm);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      printf("res_norm = %lf\n", res_norm);
#endif

      ratio = res_norm/init_norm;
#ifdef ENABLE_TESTS
      printf("ratio = %lf\n", ratio);
#endif

      if (iter > 0) {
        // A*x=r
        hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, x->desc, &one, r_vec->desc, PRECI_CUDA, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
        hipMalloc(&buff, bufferSizeMV);
        hipsparseSpMV(*handle,
              HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
              &n_one,//alpha
              A->desc,//matrix
              x->desc,//vector
              &one,//beta
              r_vec->desc,//answer
              PRECI_CUDA,//data type
              HIPSPARSE_MV_ALG_DEFAULT,//algorithm
              buff//buffer
              );
//        hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif
      //r = b - r
        hipblasDaxpy(*handle_blas, n, &ne_one, b->val, 1, r_vec->val, 1);
  //      hipDeviceSynchronize();
        hipblasDscal(*handle_blas, n, &ne_one, r_vec->val, 1);
    //    hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif
      }

//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      int error = hipGetLastError();
      printf("%s - %s\n", hipGetErrorName(error), hipGetErrorString(error));
#endif
    /*
      hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("\nend of iteration %d\n x1 = %lf \t alpha= %lf \t beta= %lf \t res_norm = %lf"
            "\n v "
            "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
            "ratio(%lf) > tolerance(%lf)\n\n\n",
            iter, onex[0], alpha, beta, res_norm, v, oner[0], onep[0], oneq[0], onez[0], ratio,
            tolerance);*/

      //printf("\e[1;1H\e[2J");
      }
#ifdef ENABLE_TESTS
    printf("TOtal of %d CuCG ITerations\n",iter);
#endif

  // free everything
    hipFree(p_vec->val);
    hipsparseDestroyDnVec(p_vec->desc);
    free(p_vec);

    hipFree(z_vec->val);
    hipsparseDestroyDnVec(z_vec->desc);
    free(z_vec);

    hipFree(q_vec->val);
    hipsparseDestroyDnVec(q_vec->desc);
    free(q_vec);

    hipFree(r_vec->val);
    hipsparseDestroyDnVec(r_vec->desc);
    free(r_vec);
    return;
}


__host__ my_cuda_csr_matrix* cusparse_crs_read(char* name)
{
  my_cuda_csr_matrix *M = (my_cuda_csr_matrix*)malloc(sizeof(my_cuda_csr_matrix));
  PRECI_DT* val;
  int* col;
  int* rowptr;

  int n = 0;
  int m = 0;
  int nz = 0;
  FILE *file;
  if ((file = fopen(name, "r"))) {
    int i;

    fscanf(file, "%d %d %d", &m, &n, &nz);

    /*PRECI_DT* val = new PRECI_DT[nz];
      int* col = new int[nz];
      int* rowptr = new int[n];*/

    val = (PRECI_DT*)malloc(sizeof(PRECI_DT)*nz);

    col = (int*)malloc(sizeof(int)*nz);
    rowptr = (int*)malloc(sizeof(int)*n+1);
    

    for (i = 0; i <= n; i++)
      fscanf(file, "%d ", &rowptr[i]);
    for (i = 0; i < nz; i++)
      fscanf(file, "%d ", &col[i]);
    for (i = 0; i < nz; i++)
      fscanf(file, PRECI_S, &val[i]);

    #ifdef ENABLE_TESTS
       printf("READ rowptr : ");
       for( i = 0; i <= n; i++)
        printf("%d ",rowptr[i]);
       printf("\n");

       printf("READ col : ");
       for( i = 0; i < nz; i++)
         printf("%d ",col[i]);
       printf("\n");

       printf("READ val : ");
       for( i = 0; i < nz; i++)
         printf("%lf ",val[i]);
       printf("\n");
    #endif

    fclose(file);
    size_t pitch;
    // Allocate memory for the CSR matrix
    //M->rowptr = hipMalloc(sizeof(int)*(n+1));
    //M->col = hipMalloc(sizeof(int)*(nz));
    //M->val = hipMalloc(sizeof(int)*(PRECI_DT));
    hipMallocPitch((void**)&M->rowptr,&pitch, (n+1) * sizeof(int),1);
    hipMallocPitch((void**)&M->col,&pitch, nz * sizeof(int),1);
    hipMallocPitch((void**)&M->val,&pitch, nz * sizeof(PRECI_DT),1);


    // Copy data from host to device
    hipMemcpy(M->rowptr, rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->col, col, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->val, val, nz * sizeof(PRECI_DT), hipMemcpyHostToDevice);

    M->n = n;
    M->m = m;
    M->nz = nz;
    hipsparseCreateCsr(&M->desc, n, n, nz, M->rowptr, M->col, M->val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, PRECI_CUDA);
    //Create the CSR matrix
   
  } else {
    printf("ERROR: could not open file %s\n", name);
    n = -1;
  }
  return M;
}


void call_CuCG(char* name, char* m_name, PRECI_DT* h_b, PRECI_DT* h_x, int maxit, PRECI_DT tol)
{
  //printf("Creating cusparse handle!\n");
  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);
  hipsparseHandle_t cusparseHandle;
  hipsparseStatus_t status = hipsparseCreate(&cusparseHandle);
  if (status != HIPSPARSE_STATUS_SUCCESS)
  {
    printf("Error creating cusparse Handle!\n"); 
  }
  else
    {
      size_t pitch;
      //printf("reading matrix file...\n");
      my_cuda_csr_matrix *A_matrix = cusparse_crs_read((char*)name);

      my_cuda_csr_matrix *M_matrix;
      if (m_name)
          M_matrix = cusparse_crs_read((char*)m_name);

      int64_t n=A_matrix->n;

      //printf("creating vectors... %d",A_matrix->n);

      // Make x vector
      my_cuda_vector *x_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
      //PRECI_DT* h_x = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
      for(int i=0;i<n;i++) h_x[i] = 0;
      hipMallocPitch((void**)&x_vec->val,&pitch, n * sizeof(PRECI_DT),1);
      hipMemcpy(x_vec->val, h_x, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);
      hipsparseCreateDnVec(&x_vec->desc, n, x_vec->val,PRECI_CUDA);

      // Make b vector
      my_cuda_vector *b_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
      //PRECI_DT* h_b = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
      //for(int i=0;i<n;i++) h_b[i] = 1;
      hipMallocPitch((void**)&b_vec->val, &pitch, n * sizeof(PRECI_DT),1);
      hipMemcpy(b_vec->val, h_b, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);
      hipsparseCreateDnVec(&b_vec->desc, n, b_vec->val,PRECI_CUDA);

      //printf("Created Vectors!\n");

      /*for (int i = 0; i < 10; i++)
	printf(PRECI_S,h_x[i]);
      printf("\n");*/

      /*for (int i = 0; i < 10; i++)
	printf(PRECI_S,h_b[i]);
      printf("\n");*/

      //printf("Calling CG func...");
      cusparse_conjugate_gradient(A_matrix, NULL, b_vec,x_vec,maxit,tol, &cusparseHandle, &cublasHandle);
      //printf("Done!\n");

      hipMemcpy(h_x, x_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(h_b, b_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);

      /*for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_x[i]);
       printf("\n");

       for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_b[i]);
      printf("\n");*/

      hipsparseDestroySpMat(A_matrix->desc);
      hipFree(A_matrix->val);
      hipFree(A_matrix->rowptr);
      hipFree(A_matrix->col);
      free(A_matrix);

      if (m_name)
{
      hipsparseDestroySpMat(M_matrix->desc);
      hipFree(M_matrix->val);
      hipFree(M_matrix->rowptr);
      hipFree(M_matrix->col);
      free(M_matrix);
}
      hipFree(x_vec->val);
      hipsparseDestroyDnVec(x_vec->desc);
      free(x_vec);

      hipFree(b_vec->val);
      hipsparseDestroyDnVec(b_vec->desc);
      free(b_vec);

      hipsparseDestroy(cusparseHandle);
      hipblasDestroy(cublasHandle);

    }
  //printf("Done!\n");

  
    
  return;
}

/*int main (void)
  {
  call_CuCG();
  return 0;
  }*/
