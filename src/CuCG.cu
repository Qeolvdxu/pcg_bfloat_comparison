#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/CuCG.h"

//#include "../include/my_crs_matrix.h"
#define PRECI_DT double 
#define PRECI_S "%lf "
#define PRECI_CUDA HIP_R_64F

typedef struct {
  hipsparseDnVecDescr_t desc;
  PRECI_DT*            val;
} my_cuda_vector;

typedef struct {
  hipsparseSpMatDescr_t desc;
  int n;
  int m;
  int nz;
  PRECI_DT *val;
  int *col;
  int *rowptr;
} my_cuda_csr_matrix;

__host__ void cusparse_conjugate_gradient(my_cuda_csr_matrix *A,
					  my_cuda_csr_matrix *M,
					  my_cuda_vector *b,
					  my_cuda_vector *x,
					  int max_iter,
			 PRECI_DT tolerance,
					  hipsparseHandle_t* handle)
{

  const double n_one = 1.0;
  const double one = 0.0;
   
  size_t bufferSizeMV;
  void* buff;
  hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, b->desc, &one, x->desc, PRECI_CUDA, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
  hipMalloc(&buff, bufferSizeMV);

  int ratio = 1;

  PRECI_DT* val = (PRECI_DT*)malloc(sizeof(PRECI_DT)*A->nz);
  hipMemcpy(val, A->val, A->nz * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  printf("%lf\n",val[0]);

  //matvec(A,x,r);
  hipsparseSpMV(*handle,
	       HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
	       &n_one,//alpha
	       A->desc,//matrix
	       b->desc,//vector
	       &one,//beta
	       x->desc,//answer
	       PRECI_CUDA,//data type
	       HIPSPARSE_MV_ALG_DEFAULT,//algorithm
	       buff//buffer
	       );
  hipDeviceSynchronize();
  auto error = hipGetLastError();
  printf("%s - %s\n", hipGetErrorName(error), hipGetErrorString(error));
  PRECI_DT* xv = (PRECI_DT*)malloc(sizeof(PRECI_DT)*A->n);
  hipMemcpy(xv, x->val, A->n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  printf("%lf\n",xv[0]);




  /*int iter = 0;
	 while (iter <= max_iter && ratio > tolerance)
	   {

	     for (int i = 0; i < 1000; i++)
	       {
		 ///matvec(A,p,q);
		 hipsparseSpMV(handle,
			      HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
			      &n_one,//alpha
			      A->desc,//matrix
			      b->desc,//vector
			      &one,//beta
			      x->desc,//answer
			      PRECI_CUDA,//data type
			      HIPSPARSE_MV_ALG_DEFAULT,//algorithm
			      buff//buffer
			      );


		 //matvec(A,x,r);
		 hipsparseSpMV(handle,
			      HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
			      &n_one,//alpha
			      A->desc,//matrix
			      x->desc,//vector
			      &one,//beta
			      b->desc,//answer
			      PRECI_CUDA,//data type
			      HIPSPARSE_MV_ALG_DEFAULT,//algorithm
			      buff//buffer
			      );



	       }
	     printf("end of iteration %d\n",iter);
	     iter++;
	     }*/

       return;
}



__host__ my_cuda_csr_matrix* cusparse_crs_read(char* name)
{
  my_cuda_csr_matrix *M = (my_cuda_csr_matrix*)malloc(sizeof(my_cuda_csr_matrix));
  PRECI_DT* val;
  int* col;
  int* rowptr;

  int n = 0;
  int m = 0;
  int nz = 0;
  FILE *file;
  if ((file = fopen(name, "r"))) {
    int i;

    fscanf(file, "%d %d %d", &m, &n, &nz);

    /*PRECI_DT* val = new PRECI_DT[nz];
      int* col = new int[nz];
      int* rowptr = new int[n];*/

    val = (PRECI_DT*)malloc(sizeof(PRECI_DT)*nz);

    col = (int*)malloc(sizeof(int)*nz);
    rowptr = (int*)malloc(sizeof(int)*n+1);
    

    for (i = 0; i <= n; i++)
      fscanf(file, "%d ", &rowptr[i]);

    for (i = 0; i < nz; i++) fscanf(file, "%d ", &col[i]);
    for (i = 0; i < nz; i++)
      fscanf(file, PRECI_S, &val[i]);

    printf("rowptr : ");


    for( i = 0; i < 5; i++)
      printf("%d, ",rowptr[i]);
    printf("\n");


    printf("col : ");

    for( i = 0; i < 5; i++)
      printf("%d, ",col[i]);
    printf("\n");

    printf("val : ");

    for( i = 0; i < 5; i++)
      printf("%lf, ",val[i]);
    printf("\n");




    fclose(file);


    // Allocate memory for the CSR matrix
    hipMalloc((void**)&M->rowptr, (n+1) * sizeof(int));
    hipMalloc((void**)&M->col, nz * sizeof(int));
    hipMalloc((void**)&M->val, nz * sizeof(PRECI_DT));


    // Copy data from host to device
    hipMemcpy(M->rowptr, rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->col, col, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->val, val, nz * sizeof(PRECI_DT), hipMemcpyHostToDevice);


    M->n = n;
    M->m = m;
    M->nz = nz;
    hipsparseCreateCsr(&M->desc, n, n, nz, M->rowptr, M->col, M->val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, PRECI_CUDA);
    //Create the CSR matrix
   
  } else {
    printf("ERROR: could not open file %s\n", name);
    n = -1;
  }
  return M;
}


void call_CuCG(void)
{
  printf("Creating cusparse handle!\n");
  hipsparseHandle_t cusparseHandle;
  hipsparseStatus_t status = hipsparseCreate(&cusparseHandle);
  if (status != HIPSPARSE_STATUS_SUCCESS)
    {
      printf("Error creating cusparse Handle!\n"); 
    }
  else
    {
      printf("reading matrix file...\n");
      my_cuda_csr_matrix *A_matrix = cusparse_crs_read((char*)"../test_subjects/rcm/bcsstk10.mtx.rcm.csr");


      int64_t n=A_matrix->n;

      printf("creating %d vectors... ",A_matrix->n);

      // Make x vector
      my_cuda_vector *x_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
      
      PRECI_DT* h_x = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
      for(int i=0;i<n;i++) h_x[i] = 1;
      hipMalloc((void**)&x_vec->val, n * sizeof(PRECI_DT));
      hipMemcpy(x_vec->val, h_x, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);
      hipsparseCreateDnVec(&x_vec->desc, n, x_vec->val,PRECI_CUDA);



      // Make b vector
      my_cuda_vector *b_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
      PRECI_DT* h_b = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
      for(int i=0;i<n;i++) h_b[i] = i;
      hipMalloc((void**)&b_vec->val, n * sizeof(PRECI_DT));
      hipMemcpy(b_vec->val, h_b, n * sizeof(PRECI_DT), hipMemcpyHostToDevice);
      hipsparseCreateDnVec(&b_vec->desc, n, b_vec->val,PRECI_CUDA);



      printf("Created Vectors!\n");  
      

      for (int i = 0; i < 10; i++)
	printf(PRECI_S,h_x[i]);
      printf("\n");


      for (int i = 0; i < 10; i++)
	printf(PRECI_S,h_b[i]);
      printf("\n");



      printf("Calling CG func...");
      cusparse_conjugate_gradient(A_matrix, A_matrix, b_vec,x_vec,8000,1e-7, &cusparseHandle);
      printf("Done!\n");

      hipMemcpy(h_x, x_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(h_b, b_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);

      for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_x[i]);
       printf("\n");



       for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_b[i]);
      printf("\n");



      hipsparseDestroy(cusparseHandle);
    }
  printf("Done!\n");

  
    
  return;
}

int main (void)
  {
  call_CuCG();
  return 0;
  }
