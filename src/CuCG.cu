#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>


#include "../include/CUSTOMIZE.h"
#include "../include/CuCG.h"


typedef struct {
  hipsparseDnVecDescr_t desc;
  CUDA_PRECI_DT_HOST*            val;
} my_cuda_vector;

typedef struct {
  hipsparseSpMatDescr_t desc;
  int n;
  int m;
  int nz;
  CUDA_PRECI_DT_HOST *val;
  int *col;
  int *rowptr;
} my_cuda_csr_matrix;

__host__ void cusparse_conjugate_gradient(my_cuda_csr_matrix *A,
					  my_cuda_csr_matrix *M,
					  my_cuda_vector *b,
					  my_cuda_vector *x,
                      my_cuda_vector *r_vec,
                      my_cuda_vector *p_vec,
                      my_cuda_vector *q_vec,
                      my_cuda_vector *z_vec,
					  int max_iter,
					 CUDA_PRECI_DT_HOST tolerance,
                      int* iter,
                      CUDA_PRECI_DT_HOST* elapsed,
                      CUDA_PRECI_DT_HOST* fault_elapsed,
					  hipsparseHandle_t* handle,
					  hipblasHandle_t* handle_blas)

{
  #ifdef ENABLE_TESTS
    printf("start cg!");
  #endif
  int n = A->n;

  CUDA_PRECI_DT_HOST* onex = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);
  #ifdef ENABLE_TESTS
  CUDA_PRECI_DT_HOST* onez = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);
  CUDA_PRECI_DT_HOST* oner = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);
  CUDA_PRECI_DT_HOST* oneq = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);
  CUDA_PRECI_DT_HOST* onep = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);
	int* rowptr;
	int* col;
	CUDA_PRECI_DT_HOST* val;
    val = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*A->nz);
    col = (int*)malloc(sizeof(int)*A->nz);
    rowptr = (int*)malloc(sizeof(int)*n+1);

   /* hipMemcpy(rowptr, M->rowptr, (A->n+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col, M->col, A->nz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(val, M->val, A->nz * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
    printf("CUDA rowptr : ");
    for( int i = 0; i <= A->n; i++)
      printf("%d ",rowptr[i]);
    printf("\n");

    printf("CUDA col : ");
    for( int i = 0; i < A->nz; i++)
      printf("%d ",col[i]);
    printf("\n");

    printf("CUDA val : ");
    for( int i = 0; i < A->nz; i++)
      printf("%lf ",val[i]);
    printf("\n");
    */
  #endif





  CUDA_PRECI_DT_HOST alpha = 1.0;
  CUDA_PRECI_DT_HOST beta = 0.0;
  const CUDA_PRECI_DT_HOST ne_one = -1.0;
  const CUDA_PRECI_DT_HOST n_one = 1.0;
  const CUDA_PRECI_DT_HOST one = 0.0;

  int itert = 0;

  CUDA_PRECI_DT_HOST v = 0;
  CUDA_PRECI_DT_HOST Rho = 0;
  CUDA_PRECI_DT_HOST Rtmp = 0;

  CUDA_PRECI_DT_HOST res_norm = 0;
  CUDA_PRECI_DT_HOST init_norm = 0;
  CUDA_PRECI_DT_HOST ratio = 0;

  
  CUDA_PRECI_DT_HOST Tiny = 0.1e-28;
  CUDA_PRECI_DT_HOST minus_alpha = 0.0;

  // x is already zero
  
  size_t bufferSizeMV;
  void* buff;
  hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc,
                          b->desc, &one, x->desc, CUDA_PRECI_DT_DEVICE, HIPSPARSE_MV_ALG_DEFAULT,
                          &bufferSizeMV);
  hipMalloc(&buff, bufferSizeMV);


  #ifdef ENABLE_TESTS
  hipMemcpy(onex, x->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  hipMemcpy(onep, p_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  hipMemcpy(oneq, q_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  hipMemcpy(oner, r_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  hipMemcpy(onez, z_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  printf("\INITIAL VEC CREATION\n x1 = %lf \t alpha= %lf \t beta= %lf "
	 "\n v "
	 "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
	 "ratio(%lf) > tolerance(%lf)\n\n\n",
	 iter, onex[0], alpha, beta, v, oner[0], onep[0], oneq[0], onez[0], ratio,
	 tolerance);
  #endif


  //matvec(A,x,r);
  hipsparseSpMV(*handle,
	       HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
	       &n_one,//alpha
	       A->desc,//matrix
	       x->desc,//vector
	       &one,//beta
	       r_vec->desc,//answer
	       CUDA_PRECI_DT_DEVICE,//data type
	       HIPSPARSE_MV_ALG_DEFAULT,//algorithm
	       buff//buffer
	       );
  //hipDeviceSynchronize();

  // r = b - r
  AXPY_FUN(*handle_blas, n, &ne_one, r_vec->val, 1, b->val, 1);
  //hipDeviceSynchronize();
  COPY_FUN(*handle_blas,n,b->val, 1, r_vec->val, 1);
  //hipDeviceSynchronize();

  // z = r
  if (M)
      //z = MT\(M\r);
     /* cusparseSPSV_solver(*handle,
                   HIPSPARSE_OPERATION_NON_TRANSPOSE,
                   &n_one,
                   M->desc,
                   r_vec->desc,
                   z_vec->desc,
                   CUDA_PRECI_DT_DEVICE,
                   HIPSPARSE_MV_ALG_DEFAULT,
                   HIPSPARSE_FILL_MODE_LOWER
                   );*/
      M = A;
  else
      // z = r
      COPY_FUN(*handle_blas,n,r_vec->val, 1, z_vec->val, 1);
  //hipDeviceSynchronize();

  // p = z
  COPY_FUN(*handle_blas,n,z_vec->val, 1, p_vec->val, 1);
  //hipDeviceSynchronize();
  NORM_FUN(*handle_blas, n, r_vec->val, 1, &res_norm);
  //hipDeviceSynchronize();
  init_norm = res_norm;
  ratio = 1.0;

  #ifdef ENABLE_TESTS
  hipMemcpy(onex, x->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  hipMemcpy(onep, p_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  hipMemcpy(oneq, q_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  hipMemcpy(oner, r_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);

  hipMemcpy(onez, z_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
  /*printf("PREQUEL \n x1 = %lf \t alpha= %lf \t beta= %lf "
	 "\n v "
	 "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
	 "ratio(%lf) > tolerance(%e)\n\n\n",
	 iter, onex[0], alpha, beta, v, oner[0], onep[0], oneq[0], onez[0], ratio,
	 tolerance);*/
  #endif
/* //CUDA_EVENT_TIME
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipEventRecord(start, stream);
  */

  // WALL TIME
  CUDA_PRECI_DT_HOST start;
  CUDA_PRECI_DT_HOST end;
  start = omp_get_wtime();

  CUDA_PRECI_DT_HOST faultcheck_start;
  CUDA_PRECI_DT_HOST faultcheck_end;

  while (itert < max_iter && ratio > tolerance)
    {
  #ifdef ENABLE_TESTS
      printf("\nITERATION %d\n",itert);
  #endif
      itert++;

      // Check X value for faults every nth iteration
      if (itert % 50 == 0)
      {
          faultcheck_start = omp_get_wtime();
          hipMemcpy(onex, x->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
          faultcheck_end = omp_get_wtime();
          *fault_elapsed += (faultcheck_end - faultcheck_start)*1000;
          //printf("faultcheck_elapsed += (%f - %f)*1000\n",(float)faultcheck_end ,(float)faultcheck_start);
      }

      if (M)
          //z = MT\(M\r);
          M=A;
      else
          // z = r
          COPY_FUN(*handle_blas,n,r_vec->val, 1, z_vec->val, 1);
      //hipDeviceSynchronize();
  #ifdef ENABLE_TESTS
      hipMemcpy(onez, z_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("z[1] = %lf\n",onez[1]);
  #endif

      // Rho = r z dot prod
      DOT_FUN(*handle_blas, n, r_vec->val, 1, z_vec->val, 1, &Rho);
      //hipDeviceSynchronize();
  #ifdef ENABLE_TESTS
      printf("Rho = %lf\n",Rho);
  #endif

      // p = z + (beta * p)
      // p = (beta * z) + p
      if (itert == 1)
	    {
	      COPY_FUN(*handle_blas,n,z_vec->val, 1, p_vec->val, 1);
//	      hipDeviceSynchronize();
	    }
      else
	    {
	      beta = Rho / (v + Tiny);
	      SCAL_FUN(*handle_blas, n, &beta, p_vec->val, 1);
	      AXPY_FUN(*handle_blas, n, &n_one, z_vec->val, 1, p_vec->val, 1);
 	//      hipDeviceSynchronize();
	    }
  #ifdef ENABLE_TESTS
	    hipMemcpy(onep, p_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
	    printf("p[1] = %lf\n",onep[1]);
  #endif
	

      hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, p_vec->desc, &one, q_vec->desc, CUDA_PRECI_DT_DEVICE, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
      hipMalloc(&buff, bufferSizeMV);

      hipsparseSpMV(*handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
        &n_one,//alpha
        A->desc,//matrix
        p_vec->desc,//vector
        &one,//beta
        q_vec->desc,//answer
        CUDA_PRECI_DT_DEVICE,//data type
        HIPSPARSE_MV_ALG_DEFAULT,//algorithm
        buff//buffer
      );
//      hipDeviceSynchronize();

#ifdef ENABLE_TESTS
      hipMemcpy(oneq, q_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("q[1] = %lf\n",oneq[1]);
#endif
      
      // Rtmp = p q dot prod
      DOT_FUN(*handle_blas, n, p_vec->val, 1, q_vec->val, 1, &Rtmp);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      printf("Rtmp = %lf\n",Rtmp);
#endif

      // v = r z dot prod
      DOT_FUN(*handle_blas, n, r_vec->val, 1, z_vec->val, 1, &v);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(onep, p_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("v = %lf\n",v);
#endif

      //alpha
      alpha = Rho / (Rtmp + Tiny);
#ifdef ENABLE_TESTS
      printf("alpha = %lf\n",alpha);
#endif
      
      // x = x + alpha * p
      AXPY_FUN(*handle_blas, n, &alpha, p_vec->val, 1, x->val, 1);
//      hipDeviceSynchronize();

#ifdef ENABLE_TESTS
      hipMemcpy(onex, x->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("x[1] = %lf\n",onex[1]);
#endif

      // r = r - alpha * q
      minus_alpha = -alpha;
      AXPY_FUN(*handle_blas, n, &minus_alpha,q_vec->val,1,r_vec->val,1);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif

      Rho = 0.0;
      NORM_FUN(*handle_blas, n, r_vec->val, 1, &res_norm);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      printf("res_norm = %lf\n", res_norm);
#endif

      ratio = res_norm/init_norm;
#ifdef ENABLE_TESTS
      printf("ratio = %lf\n", ratio);
#endif

      if (itert > 1){
        // A*x=r TONIGHTS_BIGGEST_LOSER
        hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, x->desc, &one, r_vec->desc, CUDA_PRECI_DT_DEVICE, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
        hipMalloc(&buff, bufferSizeMV);
        hipsparseSpMV(*handle,
              HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
              &n_one,//alpha
              A->desc,//matrix
              x->desc,//vector
              &one,//beta
              r_vec->desc,//answer
              CUDA_PRECI_DT_DEVICE,//data type
              HIPSPARSE_MV_ALG_DEFAULT,//algorithm
              buff//buffer
              );
//        hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif
      //r = b - r
        AXPY_FUN(*handle_blas, n, &ne_one, b->val, 1, r_vec->val, 1);
  //      hipDeviceSynchronize();
        SCAL_FUN(*handle_blas, n, &ne_one, r_vec->val, 1);
    //    hipDeviceSynchronize();
        }
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif

//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      int error = hipGetLastError();
      printf("%s - %s\n", hipGetErrorName(error), hipGetErrorString(error));
#endif
    /*
      hipMemcpy(onex, x->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      hipMemcpy(onep, p_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      hipMemcpy(oneq, q_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      hipMemcpy(oner, r_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      hipMemcpy(onez, z_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      printf("\nend of iteration %d\n x1 = %lf \t alpha= %lf \t beta= %lf \t res_norm = %lf"
            "\n v "
            "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
            "ratio(%lf) > tolerance(%lf)\n\n\n",
            iter, onex[0], alpha, beta, res_norm, v, oner[0], onep[0], oneq[0], onez[0], ratio,
            tolerance);*/

      //printf("\e[1;1H\e[2J");
      }
  hipDeviceSynchronize();

  // WALL TIME
  end = omp_get_wtime();
/* //CUDA EVENT TIME
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
*/
#ifdef ENABLE_TESTS
    printf("TOtal of %d CuCG ITerations\n",itert);
#endif

    printf(" * fault check elapsed: %f \n",(float)*fault_elapsed);
    printf(" * total elapsed: %f \n",(float)*elapsed);
    *elapsed = (end - start)*1000;
    *iter = itert;
    return;
}


__host__ my_cuda_csr_matrix* cusparse_crs_read(char* name)
{
  my_cuda_csr_matrix *M = (my_cuda_csr_matrix*)malloc(sizeof(my_cuda_csr_matrix));
  CUDA_PRECI_DT_HOST* val;
  int* col;
  int* rowptr;

  int n = 0;
  int m = 0;
  int nz = 0;
  FILE *file;
  if ((file = fopen(name, "r"))) {
    int i;


      if (fscanf(file, "%d %d %d", &m, &n, &nz) < 0)
      {
          printf("error scanning head file %s\n",name);
      }

    /*CUDA_PRECI_DT_HOST* val = new CUDA_PRECI_DT_HOST[nz];
      int* col = new int[nz];
      int* rowptr = new int[n];*/

    val = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*nz);

    col = (int*)malloc(sizeof(int)*nz);
    rowptr = (int*)malloc(sizeof(int)*n+1);


    for (i = 0; i <= n; i++)
    {
      if(fscanf(file, "%d ", &rowptr[i]) < 0)
      {
        printf("error scanning rowptr file %s\n",name);
        break;
      }
    }
    for (i = 0; i < nz; i++)
    {
      if(fscanf(file, "%d ", &col[i]) < 0)
      {
        printf("error scanning col file %s\n",name);
        break;
      }
    }
    for (i = 0; i < nz; i++)
    {
      if(fscanf(file, CUDA_PRECI_S, &val[i]) < 0)
      {
        printf("error scanning val file %s\n",name);
        break;
      }
    }

    #ifdef ENABLE_TESTS
       printf("READ rowptr : ");
       for( i = 0; i <= n; i++)
        printf("%d ",rowptr[i]);
       printf("\n");

       printf("READ col : ");
       for( i = 0; i < nz; i++)
         printf("%d ",col[i]);
       printf("\n");

       printf("READ val : ");
       for( i = 0; i < nz; i++)
         printf(CUDA_PRECI_S,val[i]);
       printf("\n");
    #endif

    fclose(file);
    //size_t pitch;

    // Allocate memory for the CSR matrix
    //M->rowptr = hipMalloc(sizeof(int)*(n+1));
    //M->col = hipMalloc(sizeof(int)*(nz));
    //M->val = hipMalloc(sizeof(int)*(CUDA_PRECI_DT_HOST));
    //hipMallocPitch((void**)&M->rowptr,&pitch, (n+1) * sizeof(int),1);
    //hipMallocPitch((void**)&M->col,&pitch, nz * sizeof(int),1);
    //hipMallocPitch((void**)&M->val,&pitch, nz * sizeof(CUDA_PRECI_DT_HOST),1);
    hipMalloc((void**)&M->rowptr, (n+1) * sizeof(int));
    hipMalloc((void**)&M->col, nz * sizeof(int));
    hipMalloc((void**)&M->val, nz * sizeof(CUDA_PRECI_DT_HOST));


    // Copy data from host to device
    hipMemcpy(M->rowptr, rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->col, col, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->val, val, nz * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);

    M->n = n;
    M->m = m;
    M->nz = nz;
    hipsparseCreateCsr(&M->desc, n, n, nz, M->rowptr, M->col, M->val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_PRECI_DT_DEVICE);
    //Create the CSR matrix

  } else {
    printf("ERROR: could not open file %s\n", name);
    n = -1;
  }
  return M;
}


void call_CuCG(char* name, char* m_name, CUDA_PRECI_DT_HOST* h_b, CUDA_PRECI_DT_HOST* h_x, int maxit,
               CUDA_PRECI_DT_HOST tol, int* iter, CUDA_PRECI_DT_HOST* elapsed, CUDA_PRECI_DT_HOST* mem_elapsed, CUDA_PRECI_DT_HOST* fault_elapsed)
{
  //printf("Creating cusparse handle!\n");
  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);
  hipsparseHandle_t cusparseHandle;
  hipsparseStatus_t status = hipsparseCreate(&cusparseHandle);

  int n = 0;
  int m = 0;
  int nz = 0;

  FILE *file;
  if (status != HIPSPARSE_STATUS_SUCCESS)
  {
    printf("Error creating cusparse Handle!\n"); 
  }
  else if ((file = fopen(name, "r")))
  {


      int i;
      if (fscanf(file, "%d %d %d", &m, &n, &nz) < 0)
      {
          printf("error scanning head file %s\n",name);
          return;
      }
      int64_t n_t=n;

  CUDA_PRECI_DT_HOST* val = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*nz);
  int* col = (int*)malloc(sizeof(int)*nz);
  int* rowptr = (int*)malloc(sizeof(int)*n+1);
  CUDA_PRECI_DT_HOST* h_rpqz = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);


    for (i = 0; i <= n; i++)
    {
      if(fscanf(file, "%d ", &rowptr[i]) < 0)
      {
        printf("error scanning rowptr file %s\n",name);
        return;
      }
    }
    for (i = 0; i < nz; i++)
    {
      if(fscanf(file, "%d ", &col[i]) < 0)
      {
        printf("error scanning col file %s\n",name);
        return;
      }
    }
    for (i = 0; i < nz; i++)
    {
      if(fscanf(file, CUDA_PRECI_S, &val[i]) < 0)
      {
        printf("error scanning val file %s\n",name);
        return;
      }
    }

    #ifdef ENABLE_TESTS
    /*  printf("READ rowptr : ");
       for( i = 0; i <= n; i++)
        printf("%d ",rowptr[i]);
       printf("\n");

       printf("READ col : ");
       for( i = 0; i < nz; i++)
         printf("%d ",col[i]);
       printf("\n");

       printf("READ val : ");
       for( i = 0; i < nz; i++)
         printf("%lf ",val[i]);
       printf("end\n");*/
    #endif
    //fclose(file);

  // Copy data from host to device


  /*EVENT TIME
   hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipEventRecord(start, stream);*/

  // WALL TIME
  CUDA_PRECI_DT_HOST start;
  CUDA_PRECI_DT_HOST end;
  start = omp_get_wtime();

  my_cuda_csr_matrix *A_matrix = (my_cuda_csr_matrix*)malloc(sizeof(my_cuda_csr_matrix));
  my_cuda_csr_matrix *M_matrix;
  my_cuda_vector *b_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *x_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *r_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *p_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *q_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *z_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  A_matrix->n = n;
  A_matrix->m = m;
  A_matrix->nz = nz;

  /*hipMallocPitch((void**)&A_matrix->rowptr,&pitch, ( n +1) * sizeof(int),1);
  hipMallocPitch((void**)&A_matrix->col,&pitch, nz * sizeof(int),1);
  hipMallocPitch((void**)&A_matrix->val,&pitch, nz * sizeof(CUDA_PRECI_DT_HOST),1);
  hipMallocPitch((void**)&x_vec->val,&pitch, n_t * sizeof(CUDA_PRECI_DT_HOST),1);
  hipMallocPitch((void**)&b_vec->val, &pitch, n_t * sizeof(CUDA_PRECI_DT_HOST),1);
  hipMallocPitch((void**)&r_vec->val,&pitch, n_t * sizeof(CUDA_PRECI_DT_HOST), 1);
  hipMallocPitch((void**)&p_vec->val,&pitch, n_t * sizeof(CUDA_PRECI_DT_HOST),1);
  hipMallocPitch((void**)&q_vec->val,&pitch, n_t * sizeof(CUDA_PRECI_DT_HOST),1);
  hipMallocPitch((void**)&z_vec->val,&pitch, n_t * sizeof(CUDA_PRECI_DT_HOST),1);*/

  hipMalloc((void**)&A_matrix->rowptr, ( n +1) * sizeof(int));
  hipMalloc((void**)&A_matrix->col, nz * sizeof(int));
  hipMalloc((void**)&A_matrix->val, nz * sizeof(CUDA_PRECI_DT_HOST));
  hipMalloc((void**)&x_vec->val, n_t * sizeof(CUDA_PRECI_DT_HOST));
  hipMalloc((void**)&b_vec->val, n_t * sizeof(CUDA_PRECI_DT_HOST));
  hipMalloc((void**)&r_vec->val, n_t * sizeof(CUDA_PRECI_DT_HOST));
  hipMalloc((void**)&p_vec->val, n_t * sizeof(CUDA_PRECI_DT_HOST));
  hipMalloc((void**)&q_vec->val, n_t * sizeof(CUDA_PRECI_DT_HOST));
  hipMalloc((void**)&z_vec->val, n_t * sizeof(CUDA_PRECI_DT_HOST));

  hipMemcpy(A_matrix->rowptr, rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(A_matrix->col, col, nz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(A_matrix->val, val, nz * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);
  hipMemcpy(x_vec->val, h_x, n_t * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);
  hipMemcpy(b_vec->val, h_b, n_t * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);
  hipMemcpy(r_vec->val, h_rpqz, n_t * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);
  hipMemcpy(p_vec->val, h_rpqz, n_t * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);
  hipMemcpy(q_vec->val, h_rpqz, n_t * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);
  hipMemcpy(z_vec->val, h_rpqz, n_t * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyHostToDevice);
  hipDeviceSynchronize();


  hipsparseCreateDnVec(&b_vec->desc, n_t, b_vec->val,CUDA_PRECI_DT_DEVICE);
  hipsparseCreateDnVec(&x_vec->desc, n_t, x_vec->val,CUDA_PRECI_DT_DEVICE);
  hipsparseCreateDnVec(&r_vec->desc, n_t, r_vec->val,CUDA_PRECI_DT_DEVICE);
  hipsparseCreateDnVec(&p_vec->desc, n_t, p_vec->val,CUDA_PRECI_DT_DEVICE);
  hipsparseCreateDnVec(&q_vec->desc, n_t, q_vec->val,CUDA_PRECI_DT_DEVICE);
  hipsparseCreateDnVec(&z_vec->desc, n_t, z_vec->val,CUDA_PRECI_DT_DEVICE);
  hipsparseCreateCsr(&A_matrix->desc, n, n, nz, A_matrix->rowptr, A_matrix->col, A_matrix->val,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, CUDA_PRECI_DT_DEVICE);


  if (m_name)
       M_matrix = cusparse_crs_read((char*)m_name);
  hipDeviceSynchronize();
  /* //EVENT TIME
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  *mem_elapsed = milliseconds;*/
   //WALL
  end = omp_get_wtime();
    *mem_elapsed = (end - start)*1000;




    //Create the CSR matrix




      //printf("creating vectors... %d",A_matrix->n);
      //CUDA_PRECI_DT_HOST* h_x = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);

      //CUDA_PRECI_DT_HOST* h_b = (CUDA_PRECI_DT_HOST*)malloc(sizeof(CUDA_PRECI_DT_HOST)*n);
      //for(int i=0;i<n;i++) h_b[i] = 1;

  // vectors
/* //WALL
    gettimeofday(&end, 0);
    CUDA_PRECI_DT_HOST seconds = end.tv_sec - begin.tv_sec;
    CUDA_PRECI_DT_HOST microseconds = end.tv_usec - begin.tv_usec;
    *elapsed = seconds + microseconds * 1e-6 * 1000;
   */

#ifdef ENABLE_TESTS
      printf("Created Vectors!\n");
     /* for (int i = 0; i < n_t; i++)
	printf(PRECI_S,h_x[i]);
      printf("\n");

      for (int i = 0; i < n_t; i++)
	printf(PRECI_S,h_b[i]);
      printf("\n");*/
      printf("Calling CG func...\n");
#endif

      cusparse_conjugate_gradient(A_matrix, NULL, b_vec,x_vec,r_vec,p_vec,q_vec,z_vec,
                                  maxit,tol, iter, elapsed, fault_elapsed, &cusparseHandle, &cublasHandle);

#ifdef ENABLE_TESTS
      printf("Done!\n");
#endif

      hipMemcpy(h_x, x_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);
      hipMemcpy(h_b, b_vec->val, n * sizeof(CUDA_PRECI_DT_HOST), hipMemcpyDeviceToHost);

#ifdef ENABLE_TESTS
     /* for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_x[i]);
       printf("\n");

       for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_b[i]);*/
      printf("\n");
#endif

      hipsparseDestroySpMat(A_matrix->desc);
      hipFree(A_matrix->val);
      hipFree(A_matrix->rowptr);
      hipFree(A_matrix->col);
      free(A_matrix);

      if (m_name)
      {
          hipsparseDestroySpMat(M_matrix->desc);
          hipFree(M_matrix->val);
          hipFree(M_matrix->rowptr);
          hipFree(M_matrix->col);
          free(M_matrix);
      }
      hipFree(x_vec->val);
      hipsparseDestroyDnVec(x_vec->desc);
      free(x_vec);

      free(h_rpqz);

      hipFree(b_vec->val);
      hipsparseDestroyDnVec(b_vec->desc);
      free(b_vec);

  // free everything
    hipFree(p_vec->val);
    hipsparseDestroyDnVec(p_vec->desc);
    free(p_vec);

    hipFree(z_vec->val);
    hipsparseDestroyDnVec(z_vec->desc);
    free(z_vec);

    hipFree(q_vec->val);
    hipsparseDestroyDnVec(q_vec->desc);
    free(q_vec);

    hipFree(r_vec->val);
    hipsparseDestroyDnVec(r_vec->desc);
    free(r_vec);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    }
  else {
    printf("ERROR: could not open file %s\n", name);
  }
  //printf("Done!\n");

  return;
}

/*int main (void)
  {
  call_CuCG();
  return 0;
  }*/
