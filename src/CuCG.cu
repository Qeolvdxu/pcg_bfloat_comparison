#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/my_crs_matrix.h"
#include "../include/CuCG.cuh"

__global__ void cgkernel()
{
  return;
}

__host__ my_crs_2_cusparse(my_crs_matrix* A, hipsparseHandle_t cusparseHandle);

{
  int nnz = A->nz;
	int n = A->n;

	// Allocate memory for the CSR matrix
	int *ptr, *indices, *data;
	hipMalloc((void **)&ptr, (n+1) * sizeof(int));
	hipMalloc((void **)&indices, nnz * sizeof(int));
	hipMalloc((void **)&data, nnz * sizeof(PRECI_DT));

	// Copy data from host to device
	hipMemcpy(ptr, A->rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(indices, A->col, nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(data, A->val, nnz * sizeof(PRECI_DT), hipMemcpyHostToDevice);

	// Create the CSR matrix
	hipsparseMatDescr_t descr;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseHybMat_t csrMat;
	hipsparseCreateHybMat(&csrMat);
	hipsparseDcsr2hyb(cusparseHandle, n, n, descr, data, ptr, indices, csrMat, 0, HIPSPARSE_HYB_PARTITION_AUTO);
}

