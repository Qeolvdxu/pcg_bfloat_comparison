#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <omp.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

#include "../include/CuCG.h"

//#include "../include/my_crs_matrix.h"
#define PRECI_DT double 
#define PRECI_S "%lf "
#define PRECI_CUDA HIP_R_64F

typedef struct {
  hipsparseDnVecDescr_t desc;
  PRECI_DT*            val;
} my_cuda_vector;

typedef struct {
  hipsparseSpMatDescr_t desc;
  int n;
  int m;
  int nz;
  PRECI_DT *val;
  int *col;
  int *rowptr;
} my_cuda_csr_matrix;

__host__ void cusparse_conjugate_gradient(my_cuda_csr_matrix *A,
					  my_cuda_csr_matrix *M,
					  my_cuda_vector *b,
					  my_cuda_vector *x,
                      my_cuda_vector *r_vec,
                      my_cuda_vector *p_vec,
                      my_cuda_vector *q_vec,
                      my_cuda_vector *z_vec,
					  int max_iter,
					 PRECI_DT tolerance,
                      int* iter,
                      double* elapsed,
					  hipsparseHandle_t* handle,
					  hipblasHandle_t* handle_blas)

{
  #ifdef ENABLE_TESTS
    printf("start cg!");
  #endif
  int n = A->n;

  #ifdef ENABLE_TESTS
  PRECI_DT* onex = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* onez = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* oner = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* oneq = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
  PRECI_DT* onep = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
	int* rowptr;
	int* col;
	PRECI_DT* val;
    val = (PRECI_DT*)malloc(sizeof(PRECI_DT)*A->nz);
    col = (int*)malloc(sizeof(int)*A->nz);
    rowptr = (int*)malloc(sizeof(int)*n+1);

   /* hipMemcpy(rowptr, M->rowptr, (A->n+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col, M->col, A->nz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(val, M->val, A->nz * sizeof(PRECI_DT), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
    printf("CUDA rowptr : ");
    for( int i = 0; i <= A->n; i++)
      printf("%d ",rowptr[i]);
    printf("\n");

    printf("CUDA col : ");
    for( int i = 0; i < A->nz; i++)
      printf("%d ",col[i]);
    printf("\n");

    printf("CUDA val : ");
    for( int i = 0; i < A->nz; i++)
      printf("%lf ",val[i]);
    printf("\n");
    */
  #endif
  size_t pitch;




  hipblasStatus_t sb;
  
  PRECI_DT alpha = 1.0;
  PRECI_DT beta = 0.0;
  const double ne_one = -1.0;
  const double n_one = 1.0;
  const double one = 0.0;

  int itert = 0;

  PRECI_DT v = 0;
  PRECI_DT Rho = 0;
  PRECI_DT Rtmp = 0;

  PRECI_DT res_norm = 0;
  PRECI_DT init_norm = 0;
  PRECI_DT ratio = 0;

  
  double Tiny = 0.1e-28;
  double minus_alpha = 0.0;

  // x is already zero
  
  size_t bufferSizeMV;
  void* buff;
  hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc,
                          b->desc, &one, x->desc, PRECI_CUDA, HIPSPARSE_MV_ALG_DEFAULT,
                          &bufferSizeMV);
  hipMalloc(&buff, bufferSizeMV);


  /*hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  printf("\INITIAL VEC CREATION\n x1 = %lf \t alpha= %lf \t beta= %lf "
	 "\n v "
	 "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
	 "ratio(%lf) > tolerance(%lf)\n\n\n",
	 iter, onex[0], alpha, beta, v, oner[0], onep[0], oneq[0], onez[0], ratio,
	 tolerance);*/


  //matvec(A,x,r);
  hipsparseSpMV(*handle,
	       HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
	       &n_one,//alpha
	       A->desc,//matrix
	       x->desc,//vector
	       &one,//beta
	       r_vec->desc,//answer
	       PRECI_CUDA,//data type
	       HIPSPARSE_MV_ALG_DEFAULT,//algorithm
	       buff//buffer
	       );
  //hipDeviceSynchronize();

  // r = b - r
  hipblasDaxpy(*handle_blas, n, &ne_one, r_vec->val, 1, b->val, 1);
  //hipDeviceSynchronize();
  hipblasDcopy(*handle_blas,n,b->val, 1, r_vec->val, 1);
  //hipDeviceSynchronize();

  // z = r
  if (M)
      //z = MT\(M\r);
      M=A;
  else
      // z = r
      hipblasDcopy(*handle_blas,n,r_vec->val, 1, z_vec->val, 1);
  //hipDeviceSynchronize();

  // p = z
  hipblasDcopy(*handle_blas,n,z_vec->val, 1, p_vec->val, 1);
  //hipDeviceSynchronize();
  hipblasDnrm2(*handle_blas, n, r_vec->val, 1, &res_norm);
  //hipDeviceSynchronize();
  init_norm = res_norm;
  ratio = 1.0;

  #ifdef ENABLE_TESTS
  hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);

  hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
  /*printf("PREQUEL \n x1 = %lf \t alpha= %lf \t beta= %lf "
	 "\n v "
	 "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
	 "ratio(%lf) > tolerance(%e)\n\n\n",
	 iter, onex[0], alpha, beta, v, oner[0], onep[0], oneq[0], onez[0], ratio,
	 tolerance);*/
  #endif
/* //CUDA_EVENT_TIME
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipEventRecord(start, stream);
  */

  // WALL TIME
  double start;
  double end;
  start = omp_get_wtime();

  while (itert <= max_iter && ratio > tolerance)
    {
  #ifdef ENABLE_TESTS
      printf("\nITERATION %d\n",itert);
  #endif
      itert++;

      if (M)
          //z = MT\(M\r);
          M=A;
      else
          // z = r
          hipblasDcopy(*handle_blas,n,r_vec->val, 1, z_vec->val, 1);
      //hipDeviceSynchronize();
  #ifdef ENABLE_TESTS
      hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("z[1] = %lf\n",onez[1]);
  #endif

      // Rho = r z dot prod
      hipblasDdot(*handle_blas, n, r_vec->val, 1, z_vec->val, 1, &Rho);
      //hipDeviceSynchronize();
  #ifdef ENABLE_TESTS
      printf("Rho = %lf\n",Rho);
  #endif

      // p = z + (beta * p)
      // p = (beta * z) + p
      if (itert == 1)
	    {
	      hipblasDcopy(*handle_blas,n,z_vec->val, 1, p_vec->val, 1);
//	      hipDeviceSynchronize();
	    }
      else
	    {
	      beta = Rho / (v + Tiny);
	      hipblasDscal(*handle_blas, n, &beta, p_vec->val, 1);
	      hipblasDaxpy(*handle_blas, n, &n_one, z_vec->val, 1, p_vec->val, 1);
 	//      hipDeviceSynchronize();
	    }
  #ifdef ENABLE_TESTS
	    printf("beta = %lf\n",beta);
	    hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
	    printf("p[1] = %lf\n",onep[1]);
  #endif
	

      hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, p_vec->desc, &one, q_vec->desc, PRECI_CUDA, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
      hipMalloc(&buff, bufferSizeMV);

      hipsparseSpMV(*handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
        &n_one,//alpha
        A->desc,//matrix
        p_vec->desc,//vector
        &one,//beta
        q_vec->desc,//answer
        PRECI_CUDA,//data type
        HIPSPARSE_MV_ALG_DEFAULT,//algorithm
        buff//buffer
      );
//      hipDeviceSynchronize();

#ifdef ENABLE_TESTS
      hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("q[1] = %lf\n",oneq[1]);
#endif
      
      // Rtmp = p q dot prod
      hipblasDdot(*handle_blas, n, p_vec->val, 1, q_vec->val, 1, &Rtmp);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      printf("Rtmp = %lf\n",Rtmp);
#endif

      // v = r z dot prod
      hipblasDdot(*handle_blas, n, r_vec->val, 1, z_vec->val, 1, &v);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("v = %lf\n",v);
#endif

      //alpha
      alpha = Rho / (Rtmp + Tiny);
#ifdef ENABLE_TESTS
      printf("alpha = %lf\n",alpha);
#endif
      
      // x = x + alpha * p
      hipblasDaxpy(*handle_blas, n, &alpha, p_vec->val, 1, x->val, 1);
//      hipDeviceSynchronize();

#ifdef ENABLE_TESTS
      hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("x[1] = %lf\n",onex[1]);
#endif

      // r = r - alpha * q
      minus_alpha = -alpha;
      hipblasDaxpy(*handle_blas, n, &minus_alpha,q_vec->val,1,r_vec->val,1);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif

      Rho = 0.0;
      hipblasDnrm2(*handle_blas, n, r_vec->val, 1, &res_norm);
//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      printf("res_norm = %lf\n", res_norm);
#endif

      ratio = res_norm/init_norm;
#ifdef ENABLE_TESTS
      printf("ratio = %lf\n", ratio);
#endif

      if (iter > 0) {
        // A*x=r
        hipsparseSpMV_bufferSize(*handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, &n_one, A->desc, x->desc, &one, r_vec->desc, PRECI_CUDA, HIPSPARSE_MV_ALG_DEFAULT, &bufferSizeMV);
        hipMalloc(&buff, bufferSizeMV);
        hipsparseSpMV(*handle,
              HIPSPARSE_OPERATION_NON_TRANSPOSE,//operation
              &n_one,//alpha
              A->desc,//matrix
              x->desc,//vector
              &one,//beta
              r_vec->desc,//answer
              PRECI_CUDA,//data type
              HIPSPARSE_MV_ALG_DEFAULT,//algorithm
              buff//buffer
              );
//        hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif
      //r = b - r
        hipblasDaxpy(*handle_blas, n, &ne_one, b->val, 1, r_vec->val, 1);
  //      hipDeviceSynchronize();
        hipblasDscal(*handle_blas, n, &ne_one, r_vec->val, 1);
    //    hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("r[1] = %lf\n", oner[1]);
#endif
      }

//      hipDeviceSynchronize();
#ifdef ENABLE_TESTS
      int error = hipGetLastError();
      printf("%s - %s\n", hipGetErrorName(error), hipGetErrorString(error));
#endif
    /*
      hipMemcpy(onex, x->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(onep, p_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(oneq, q_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(oner, r_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(onez, z_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      printf("\nend of iteration %d\n x1 = %lf \t alpha= %lf \t beta= %lf \t res_norm = %lf"
            "\n v "
            "= %lf\nr0 = %lf \n p0 = %lf\n q0 = %lf\n z0 = %lf\n if (norm "
            "ratio(%lf) > tolerance(%lf)\n\n\n",
            iter, onex[0], alpha, beta, res_norm, v, oner[0], onep[0], oneq[0], onez[0], ratio,
            tolerance);*/

      //printf("\e[1;1H\e[2J");
      }
  hipDeviceSynchronize();

  // WALL TIME
  end = omp_get_wtime();
/* //CUDA EVENT TIME
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
*/
#ifdef ENABLE_TESTS
    printf("TOtal of %d CuCG ITerations\n",itert);
#endif

    *elapsed = (end - start)*1000;
    *iter = itert;
    return;
}


__host__ my_cuda_csr_matrix* cusparse_crs_read(char* name)
{
  my_cuda_csr_matrix *M = (my_cuda_csr_matrix*)malloc(sizeof(my_cuda_csr_matrix));
  PRECI_DT* val;
  int* col;
  int* rowptr;

  int n = 0;
  int m = 0;
  int nz = 0;
  FILE *file;
  if ((file = fopen(name, "r"))) {
    int i;

    fscanf(file, "%d %d %d", &m, &n, &nz);

    /*PRECI_DT* val = new PRECI_DT[nz];
      int* col = new int[nz];
      int* rowptr = new int[n];*/

    val = (PRECI_DT*)malloc(sizeof(PRECI_DT)*nz);

    col = (int*)malloc(sizeof(int)*nz);
    rowptr = (int*)malloc(sizeof(int)*n+1);


    for (i = 0; i <= n; i++)
      fscanf(file, "%d ", &rowptr[i]);
    for (i = 0; i < nz; i++)
      fscanf(file, "%d ", &col[i]);
    for (i = 0; i < nz; i++)
      fscanf(file, PRECI_S, &val[i]);

    #ifdef ENABLE_TESTS
       printf("READ rowptr : ");
       for( i = 0; i <= n; i++)
        printf("%d ",rowptr[i]);
       printf("\n");

       printf("READ col : ");
       for( i = 0; i < nz; i++)
         printf("%d ",col[i]);
       printf("\n");

       printf("READ val : ");
       for( i = 0; i < nz; i++)
         printf("%lf ",val[i]);
       printf("\n");
    #endif

    fclose(file);
    size_t pitch;
    // Allocate memory for the CSR matrix
    //M->rowptr = hipMalloc(sizeof(int)*(n+1));
    //M->col = hipMalloc(sizeof(int)*(nz));
    //M->val = hipMalloc(sizeof(int)*(PRECI_DT));
    hipMallocPitch((void**)&M->rowptr,&pitch, (n+1) * sizeof(int),1);
    hipMallocPitch((void**)&M->col,&pitch, nz * sizeof(int),1);
    hipMallocPitch((void**)&M->val,&pitch, nz * sizeof(PRECI_DT),1);


    // Copy data from host to device
    hipMemcpy(M->rowptr, rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->col, col, nz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(M->val, val, nz * sizeof(PRECI_DT), hipMemcpyHostToDevice);

    M->n = n;
    M->m = m;
    M->nz = nz;
    hipsparseCreateCsr(&M->desc, n, n, nz, M->rowptr, M->col, M->val, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, PRECI_CUDA);
    //Create the CSR matrix

  } else {
    printf("ERROR: could not open file %s\n", name);
    n = -1;
  }
  return M;
}


void call_CuCG(char* name, char* m_name, PRECI_DT* h_b, PRECI_DT* h_x, int maxit,
               PRECI_DT tol, int* iter, double* elapsed, double* mem_elapsed)
{
  //printf("Creating cusparse handle!\n");
  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);
  hipsparseHandle_t cusparseHandle;
  hipsparseStatus_t status = hipsparseCreate(&cusparseHandle);

  int n = 0;
  int m = 0;
  int nz = 0;

  FILE *file;
  if (status != HIPSPARSE_STATUS_SUCCESS)
  {
    printf("Error creating cusparse Handle!\n"); 
  }
  else if ((file = fopen(name, "r")))
  {
      size_t pitch;


      int i;
      fscanf(file, "%d %d %d", &m, &n, &nz);
      int64_t n_t=n;

  PRECI_DT* val = (PRECI_DT*)malloc(sizeof(PRECI_DT)*nz);
  int* col = (int*)malloc(sizeof(int)*nz);
  int* rowptr = (int*)malloc(sizeof(int)*n+1);
  PRECI_DT* h_rpqz = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);


    for (i = 0; i <= n; i++)
      fscanf(file, "%d ", &rowptr[i]);
    for (i = 0; i < nz; i++)
      fscanf(file, "%d ", &col[i]);
    for (i = 0; i < nz; i++)
      fscanf(file, PRECI_S, &val[i]);

    #ifdef ENABLE_TESTS
      printf("READ rowptr : ");
       for( i = 0; i <= n; i++)
        printf("%d ",rowptr[i]);
       printf("\n");

       printf("READ col : ");
       for( i = 0; i < nz; i++)
         printf("%d ",col[i]);
       printf("\n");

       printf("READ val : ");
       for( i = 0; i < nz; i++)
         printf("%lf ",val[i]);
       printf("end\n");
    #endif
    //fclose(file);

  // Copy data from host to device


  /*EVENT TIME
   hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipEventRecord(start, stream);*/

  // WALL TIME
  double start;
  double end;
  start = omp_get_wtime();

  my_cuda_csr_matrix *A_matrix = (my_cuda_csr_matrix*)malloc(sizeof(my_cuda_csr_matrix));
  my_cuda_vector *b_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *x_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *r_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *p_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *q_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_vector *z_vec = (my_cuda_vector*)malloc(sizeof(my_cuda_vector));
  my_cuda_csr_matrix *M_matrix;
  A_matrix->n = n;
  A_matrix->m = m;
  A_matrix->nz = nz;

  hipMallocPitch((void**)&A_matrix->rowptr,&pitch, ( n +1) * sizeof(int),1);
  hipMallocPitch((void**)&A_matrix->col,&pitch, nz * sizeof(int),1);
  hipMallocPitch((void**)&A_matrix->val,&pitch, nz * sizeof(PRECI_DT),1);
  hipMallocPitch((void**)&x_vec->val,&pitch, n_t * sizeof(PRECI_DT),1);
  hipMallocPitch((void**)&b_vec->val, &pitch, n_t * sizeof(PRECI_DT),1);
  hipMallocPitch((void**)&r_vec->val,&pitch, n_t * sizeof(PRECI_DT), 1);
  hipMallocPitch((void**)&p_vec->val,&pitch, n_t * sizeof(PRECI_DT),1);
  hipMallocPitch((void**)&q_vec->val,&pitch, n_t * sizeof(PRECI_DT),1);
  hipMallocPitch((void**)&z_vec->val,&pitch, n_t * sizeof(PRECI_DT),1);

  hipMemcpy(A_matrix->rowptr, rowptr, (n+1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(A_matrix->col, col, nz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(A_matrix->val, val, nz * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(x_vec->val, h_x, n_t * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(b_vec->val, h_b, n_t * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(r_vec->val, h_rpqz, n_t * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(p_vec->val, h_rpqz, n_t * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(q_vec->val, h_rpqz, n_t * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipMemcpy(z_vec->val, h_rpqz, n_t * sizeof(PRECI_DT), hipMemcpyHostToDevice);
  hipDeviceSynchronize();


  hipsparseCreateDnVec(&b_vec->desc, n_t, b_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&x_vec->desc, n_t, x_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&r_vec->desc, n_t, r_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&p_vec->desc, n_t, p_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&q_vec->desc, n_t, q_vec->val,PRECI_CUDA);
  hipsparseCreateDnVec(&z_vec->desc, n_t, z_vec->val,PRECI_CUDA);
  hipsparseCreateCsr(&A_matrix->desc, n, n, nz, A_matrix->rowptr, A_matrix->col, A_matrix->val,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, PRECI_CUDA);
  hipDeviceSynchronize();
  /* //EVENT TIME
  hipEventRecord(stop, stream);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  *mem_elapsed = milliseconds;*/
   //WALL
  end = omp_get_wtime();
    *mem_elapsed = (end - start)*1000;




    //Create the CSR matrix


   if (m_name)
       M_matrix = cusparse_crs_read((char*)m_name);


      //printf("creating vectors... %d",A_matrix->n);
      //PRECI_DT* h_x = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);

      //PRECI_DT* h_b = (PRECI_DT*)malloc(sizeof(PRECI_DT)*n);
      //for(int i=0;i<n;i++) h_b[i] = 1;

  // vectors
/* //WALL
    gettimeofday(&end, 0);
    double seconds = end.tv_sec - begin.tv_sec;
    double microseconds = end.tv_usec - begin.tv_usec;
    *elapsed = seconds + microseconds * 1e-6 * 1000;
   */

#ifdef ENABLE_TESTS
      printf("Created Vectors!\n");
     /* for (int i = 0; i < n_t; i++)
	printf(PRECI_S,h_x[i]);
      printf("\n");

      for (int i = 0; i < n_t; i++)
	printf(PRECI_S,h_b[i]);
      printf("\n");*/
      printf("Calling CG func...\n");
#endif

      cusparse_conjugate_gradient(A_matrix, NULL, b_vec,x_vec,r_vec,p_vec,q_vec,z_vec,
                                  maxit,tol, iter, elapsed, &cusparseHandle, &cublasHandle);

#ifdef ENABLE_TESTS
      printf("Done!\n");
#endif

      hipMemcpy(h_x, x_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);
      hipMemcpy(h_b, b_vec->val, n * sizeof(PRECI_DT), hipMemcpyDeviceToHost);

#ifdef ENABLE_TESTS
     /* for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_x[i]);
       printf("\n");

       for (int i = 0; i < 10; i++)
	 printf(PRECI_S,h_b[i]);*/
      printf("\n");
#endif

      hipsparseDestroySpMat(A_matrix->desc);
      hipFree(A_matrix->val);
      hipFree(A_matrix->rowptr);
      hipFree(A_matrix->col);
      free(A_matrix);

      if (m_name)
      {
          hipsparseDestroySpMat(M_matrix->desc);
          hipFree(M_matrix->val);
          hipFree(M_matrix->rowptr);
          hipFree(M_matrix->col);
          free(M_matrix);
      }
      hipFree(x_vec->val);
      hipsparseDestroyDnVec(x_vec->desc);
      free(x_vec);

      free(h_rpqz);

      hipFree(b_vec->val);
      hipsparseDestroyDnVec(b_vec->desc);
      free(b_vec);

  // free everything
    hipFree(p_vec->val);
    hipsparseDestroyDnVec(p_vec->desc);
    free(p_vec);

    hipFree(z_vec->val);
    hipsparseDestroyDnVec(z_vec->desc);
    free(z_vec);

    hipFree(q_vec->val);
    hipsparseDestroyDnVec(q_vec->desc);
    free(q_vec);

    hipFree(r_vec->val);
    hipsparseDestroyDnVec(r_vec->desc);
    free(r_vec);
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    }
  else {
    printf("ERROR: could not open file %s\n", name);
  }
  //printf("Done!\n");

  return;
}

/*int main (void)
  {
  call_CuCG();
  return 0;
  }*/
